#include "hip/hip_runtime.h"
// This file is part of the PSEv3 plugin, released under the BSD 3-Clause License
//
// Andrew Fiore

#include "Helper_Integrator.cuh"

#include "hoomd/RNGIdentifiers.h"
#include "hoomd/RandomNumbers.h"

#include "hoomd/TextureTools.h"


#include <stdio.h>
#include <math.h>

#include "lapacke.h"
#include "cblas.h"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif



/*! 
	Helper_Integrator.cu

	Helper functions for saddle point integration
*/
	
/*!
  	Generate random numbers on particles.
	
	d_psi		(output) random vector
        n		(input)  number of particles
	timestep	(input)  length of time step
	seed		(input)  seed for random number generation

*/

namespace hoomd
{
namespace md
{

__global__ void Integrator_RFD_RandDisp_kernel(
								float *d_psi,
								unsigned int N,
								
								// Edmond 03/31/2023 : rand number now passed in seed & timestep
								// const unsigned int seed
								uint64_t timestep,
								uint16_t seed
								){

	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	// Check if thread is in bounds
	if (idx < N) {

		// // Initialize random seed
        //         detail::Saru s(idx, seed);
		// Edmond 03/31/2023:
		// uint16_t seed = m_sysdef->getSeed();
		// Initialize the RNG
		RandomGenerator rng(hoomd::Seed(50, timestep, seed),
			hoomd::Counter(idx));
		// Square root of 3
		float sqrt3 = 1.732050807568877;
		
		// Call the random number generator
		// Edmond 03/31/2023:
		hoomd::UniformDistribution<Scalar> uniform(Scalar(-sqrt3), Scalar(sqrt3));
		float x1 = uniform(rng);
		float y1 = uniform(rng);
		float z1 = uniform(rng);
		float x2 = uniform(rng);
		float y2 = uniform(rng);
		float z2 = uniform(rng);

		// Write to output
		d_psi[ 6*idx + 0 ] = x1;
		d_psi[ 6*idx + 1 ] = y1;
		d_psi[ 6*idx + 2 ] = z1;
		d_psi[ 6*idx + 3 ] = x2;
		d_psi[ 6*idx + 4 ] = y2;
		d_psi[ 6*idx + 5 ] = z2; 

	}

}

/*! 
	The output velocity

	d_b	(output) output vector
   	N 	(input)  number of particles

*/
__global__ void Integrator_ZeroVelocity_kernel( 
						float *d_b,
						unsigned int N
						){

	// Thread index
	unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
	
	// Check if thread is inbounds
	if ( tid < N ) {
	
		d_b[ 6*tid + 0 ] = 0.0;
		d_b[ 6*tid + 1 ] = 0.0;
		d_b[ 6*tid + 2 ] = 0.0;
		d_b[ 6*tid + 3 ] = 0.0;
		d_b[ 6*tid + 4 ] = 0.0;
		d_b[ 6*tid + 5 ] = 0.0;
	
	}
}

/*! 
	Add rate of strain from shearing to the right-hand side of the saddle point solve

	d_b		(input/output) 	right-hand side vector
	shear_rate 	(input) 	shear rate of applied deformation
   	N 		(input)  	number of particles

*/
__global__ void Integrator_AddStrainRate_kernel( 
						float *d_b,
						float shear_rate,
						unsigned int N
						){

	// Thread index
	unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
	
	// Check if thread is inbounds
	if ( tid < N ) {


		// Index into array
		int ind = 6*N + 5*tid;

		// Add strain rate. For each particle, stores
		// [ F1, F2, F3, L1, L2, L3, E1, E2, E3, E4, E5 ]	
		d_b[ ind + 0 ] += 0.0;
		d_b[ ind + 1 ] += shear_rate; //zhoge: because it is 2E_xy, see "Computational tricks"
		d_b[ ind + 2 ] += 0.0;
		d_b[ ind + 3 ] += 0.0;
		d_b[ ind + 4 ] += 0.0;

	}
}

}	// end namespace md
}	// end namespace hoomd